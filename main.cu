#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// #include <hip/hip_runtime.h>

#define MATCH 1
#define MISMATCH -1
#define GAP -2

#define NUM_THREADS 256

__global__ void NW_kernel(int* d_data, int* d_dataComputed, const char* sequence_a, const char* sequence_b){
    // get index
    for( int index = blockIdx.x * blockDim.x + threadIdx.x; index < rows*cols; index += blockDim.x*gridDim.x ){
        int y = floor(index/(double)rows);
        int x = index % cols;
        if((y == 0 || x == 0) || (y > rows || x > cols)) continue;


        int top_index = index - cols;
        int left_index = index-1;
        int diagonal_index = top_index-1;

        while( !d_dataComputed[top_index] || !d_dataComputed[left_index] || !d_dataComputed[diagonal_index] ) { continue; } // wait for values to be filled in d_data

        int top_score = d_data[top_index] * GAP;
        int left_score = d_data[left_index] * GAP;
        int diagonal_score = sequence_a[x] == sequence_b[x] ? d_data[diagonal_index] * MATCH : d_data[diagonal_index] * MISMATCH;

        d_dataComputed[index] = 1;
    }

}


void NW_kernelLaunch( int* d_data, int* d_dataComputed, int rows, int cols, int grid_length, char sequence_a, char sequence_b ){
    // controller for thread launching
    int diagonals = grid_length - 1;

    // #ifdef DIAGONAL_SERIAL
    // // for each diagonal launch a kernel
    // for(int i = 0; i < diagonals; i++){
    //     NW_kernel<<<1, NUM_THREADS>>>(d_data, d_dataComputed, seqeunce_a, sequence_b, rows, cols);

    //     hipDeviceSynchronize(); // sync before moving onto next diagonal
    // }
    // #endif

    #ifdef DIAGONAL_PARAALLEL
    NW_kernel<<<diagonals, NUM_THREADS>>>(d_data, sequence_a, sequence_b, rows, cols);
    #endif
}

static inline int min(int a, int b) { return a < b ? a : b; }
static inline int max(int a, int b) { return a > b ? a : b; }

static inline void NW_init(int* d_data, int* d_dataComputed, int rows, int cols){
    int diff_min = min(rows, cols);
    int diff_max = max(rows, cols) - diff_min;

    d_data[0] = 0;
    d_dataComputed[0] = 1;


    // o(n) despite two loops :)
    int i;
    for(i = 1; i < diff_min; i++){
        // d_data[0][i] = d_data[0][i-1]-1;
        // d_data[i][0] = d_data[i-1][0]-1;
        int row_index = i*cols;

        printf("row_index %d, cols %d\n", row_index, cols);
        d_data[row_index] = d_data[row_index-cols] - 1; // rows
        d_data[i] = d_data[i-1] - 1; //cols

        d_dataComputed[row_index] = 1;
        d_dataComputed[i] = 0;
    }

    if (rows == cols) return;

    int rows_bigger = 0;
    if(rows > cols) rows_bigger = 1;

    for(; i < diff_max; i++){
        if(rows_bigger) {
            int row_index = i*cols;

            d_data[i*cols] = d_data[row_index-cols] - 1; // rows
            d_dataComputed[row_index] = 1;
        }
        else{
            d_data[i] = d_data[i-1] - 1; // cols
            d_dataComputed[i] = 1;
        }
    }
}

// needleman-wunsch algorithm
int main(int argc, char* argv[]){
    if(argc > 1){
        printf("Incorrect number of inputs\n");
        return EXIT_FAILURE;
    }

    // test sequences
    char* sequence_a = "GCATGCG";
    char* sequence_b = "GATTACA";

    // construct a 1d array to represent the 2d grid for needleman
    int grid_length = (strlen(sequence_a)+1) + (strlen(sequence_b)+1);
    int cols = strlen(sequence_a)+1;
    int rows = strlen(sequence_b)+1;

    int* d_data = calloc(grid_length, sizeof(int));
    int* d_dataComputed = calloc(grid_length, sizeof(int));

    NW_init(d_data, d_dataComputed, rows, cols);

    for(size_t i = 0; i < rows; i++){
        for(size_t j = 0; j < cols; j++){
            int index = i*cols + j;
            printf("%d ", d_data[index]);
        }
        printf("\n");
    }

    NW_kernelLaunch( d_data, d_dataComputed, rows, cols, grid_length, sequence_a, sequence_b );


    return EXIT_SUCCESS;
}
